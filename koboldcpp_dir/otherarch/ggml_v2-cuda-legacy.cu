#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <stdint.h>
#include <stdio.h>
#include <atomic>

#if defined(GGML_USE_HIPBLAS)
#include <hip/hip_runtime.h>
#include <hipblas/hipblas.h>
#include <hip/hip_fp16.h>
#define HIPBLAS_COMPUTE_32F HIPBLAS_R_32F
#define HIPBLAS_COMPUTE_32F_FAST_16F HIPBLAS_R_32F
#define HIPBLAS_GEMM_DEFAULT HIPBLAS_GEMM_DEFAULT
#define HIPBLAS_OP_N HIPBLAS_OP_N
#define HIPBLAS_OP_T HIPBLAS_OP_T
#define HIPBLAS_STATUS_SUCCESS HIPBLAS_STATUS_SUCCESS
#define HIPBLAS_TF32_TENSOR_OP_MATH 0
#define HIP_R_16F  HIPBLAS_R_16F
#define HIP_R_32F  HIPBLAS_R_32F
#define __shfl_xor_sync(mask, var, laneMask, width) __shfl_xor(var, laneMask, width)
#define hipblasCreate hipblasCreate
#define hipblasGemmEx hipblasGemmEx
#define hipblasHandle_t hipblasHandle_t
#define hipblasSetMathMode(handle, mode) HIPBLAS_STATUS_SUCCESS
#define hipblasSetStream hipblasSetStream
#define hipblasSgemm hipblasSgemm
#define hipblasStatus_t hipblasStatus_t
#define hipDeviceProp_t hipDeviceProp_t
#define hipDeviceSynchronize hipDeviceSynchronize
#define hipError_t hipError_t
#define hipEventCreateWithFlags hipEventCreateWithFlags
#define hipEventDisableTiming hipEventDisableTiming
#define hipEventRecord hipEventRecord
#define hipEvent_t hipEvent_t
#define hipFree hipFree
#define hipHostFree hipHostFree
#define hipGetDevice hipGetDevice
#define hipGetDeviceCount hipGetDeviceCount
#define hipGetDeviceProperties hipGetDeviceProperties
#define hipGetErrorString hipGetErrorString
#define hipGetLastError hipGetLastError
#define hipMalloc hipMalloc
#define hipHostMalloc(ptr, size) hipHostMalloc(ptr, size, hipHostMallocDefault)
#define hipMemcpy hipMemcpy
#define hipMemcpy2DAsync hipMemcpy2DAsync
#define hipMemcpyAsync hipMemcpyAsync
#define hipMemcpyDeviceToDevice hipMemcpyDeviceToDevice
#define hipMemcpyDeviceToHost hipMemcpyDeviceToHost
#define hipMemcpyHostToDevice hipMemcpyHostToDevice
#define hipMemcpyKind hipMemcpyKind
#define hipMemset hipMemset
#define hipOccupancyMaxPotentialBlockSize hipOccupancyMaxPotentialBlockSize
#define hipSetDevice hipSetDevice
#define hipStreamCreateWithFlags hipStreamCreateWithFlags
#define hipStreamNonBlocking hipStreamNonBlocking
#define hipStreamSynchronize hipStreamSynchronize
#define hipStreamWaitEvent hipStreamWaitEvent
#define hipStream_t hipStream_t
#define hipSuccess hipSuccess
#else
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#endif

#include "ggml_v2-cuda-legacy.h"
#include "ggml_v2-cuda.h"
#include "ggml_v2.h"

static_assert(sizeof(half) == sizeof(ggml_v2_fp16_t), "wrong fp16 size");

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "cuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);    \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

typedef void (*to_fp32_cuda_t)(const void * x, float * y, int k, hipStream_t stream);

#define QK4_0 32
typedef struct {
    float   d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(float) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
typedef struct {
    float   d;              // delta
    float   m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(float) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK4_2 16
typedef struct {
    half  d;                // delta
    uint8_t qs[QK4_2 / 2];  // nibbles / quants
} block_q4_2;
static_assert(sizeof(block_q4_2) == sizeof(ggml_v2_fp16_t) + QK4_2 / 2, "wrong q4_2 block size/padding");

#define QK4_3 16
typedef struct {
    __half  d;              // delta
    __half  m;              // min
    uint8_t qs[QK4_3 / 2];  // nibbles / quants
} block_q4_3;
static_assert(sizeof(block_q4_3) == 2 * sizeof(ggml_v2_fp16_t) + QK4_3 / 2, "wrong q4_3 block size/padding");

#define QK5_0 32
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_v2_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
typedef struct {
    half d;                 // delta
    half m;                 // min
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_v2_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
typedef struct {
    float   d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(float) + QK8_0, "wrong q8_0 block size/padding");

static __global__ void dequantize_block_q4_0(const void * vx, float * y) {
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_0 + l + 0] = v0;
        y[i*QK4_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_1(const void * vx, float * y) {
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK4_1 + l + 0] = v0;
        y[i*QK4_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_2(const void * vx, float * y) {
    const block_q4_2 * x = (const block_q4_2 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_2; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_2 + l + 0] = v0;
        y[i*QK4_2 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_3(const void * vx, float * y) {
    const block_q4_3 * x = (const block_q4_3 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_3; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK4_3 + l + 0] = v0;
        y[i*QK4_3 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q5_0(const void * vx, float * y) {
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    uint32_t qh;
    memcpy(&qh, x[i].qh, sizeof(qh));

    for (int l = 0; l < QK5_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vh0 = ((qh & (1 << (l + 0))) >> (l + 0)) << 4;
        const int8_t vh1 = ((qh & (1 << (l + 1))) >> (l + 1)) << 4;

        const int8_t vi0 = ((vi & 0xf) | vh0);
        const int8_t vi1 = ((vi >>  4) | vh1);

        const float v0 = (vi0 - 16)*d;
        const float v1 = (vi1 - 16)*d;

        y[i*QK5_0 + l + 0] = v0;
        y[i*QK5_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q5_1(const void * vx, float * y) {
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    uint32_t qh;
    memcpy(&qh, x[i].qh, sizeof(qh));

    for (int l = 0; l < QK5_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vh0 = ((qh & (1 << (l + 0))) >> (l + 0)) << 4;
        const int8_t vh1 = ((qh & (1 << (l + 1))) >> (l + 1)) << 4;

        const int8_t vi0 = (vi & 0xf) | vh0;
        const int8_t vi1 = (vi >>  4) | vh1;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK5_1 + l + 0] = v0;
        y[i*QK5_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q8_0(const void * vx, float * y) {
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const int8_t * pp = x[i].qs;

    for (int l = 0; l < QK8_0; l++) {
        const int8_t vi = pp[l];

        y[i*QK8_0 + l] = vi*d;
    }
}

static void dequantize_row_q4_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_0;
    dequantize_block_q4_0<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q4_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_1;
    dequantize_block_q4_1<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q4_2_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_2;
    dequantize_block_q4_2<<<nb, 1, 0, stream>>>(vx, y);
}

void dequantize_row_q4_3_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_3;
    dequantize_block_q4_3<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_0;
    dequantize_block_q5_0<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_1;
    dequantize_block_q5_1<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q8_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK8_0;
    dequantize_block_q8_0<<<nb, 1, 0, stream>>>(vx, y);
}

// TODO: optimize
static __global__ void convert_fp16_to_fp32(const void * vx, float * y) {
    const half * x = (const half *) vx;

    const int i = blockIdx.x;

    y[i] = __half2float(x[i]);
}

static void convert_fp16_to_fp32_cuda(const void * x, float * y, int k, hipStream_t stream) {
    convert_fp16_to_fp32<<<k, 1, 0, stream>>>(x, y);
}

static to_fp32_cuda_t ggml_v2_get_to_fp32_cuda(ggml_v2_type type) {
    switch (type) {
        case GGML_V2_TYPE_Q4_0:
            return dequantize_row_q4_0_cuda;
        case GGML_V2_TYPE_Q4_1:
            return dequantize_row_q4_1_cuda;
        case GGML_V2_TYPE_Q4_2:
            return dequantize_row_q4_2_cuda;
        case GGML_V2_TYPE_Q4_3:
            return dequantize_row_q4_3_cuda;
        case GGML_V2_TYPE_Q5_0:
            return dequantize_row_q5_0_cuda;
        case GGML_V2_TYPE_Q5_1:
            return dequantize_row_q5_1_cuda;
        case GGML_V2_TYPE_Q8_0:
            return dequantize_row_q8_0_cuda;
        case GGML_V2_TYPE_F16:
            return convert_fp16_to_fp32_cuda;
        default:
            return nullptr;
    }
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS_V2 16

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
};

static cuda_buffer g_cuda_buffer_pool[MAX_CUDA_BUFFERS_V2];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

static void * ggml_v2_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS_V2; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}

static void ggml_v2_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS_V2; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS_V2\n");
    CUDA_CHECK(hipFree(ptr));
}

#define GGML_V2_CUDA_MAX_STREAMS 8 // Set this to 1 for reproducible matrix multiplication.
#define GGML_V2_CUDA_MAX_EVENTS 64
static hipblasHandle_t g_cublasH = nullptr;
static hipStream_t g_cudaStreams[GGML_V2_CUDA_MAX_STREAMS] = { nullptr };
static hipStream_t g_cudaStreams2[GGML_V2_CUDA_MAX_STREAMS] = { nullptr };
static hipEvent_t g_cudaEvents[GGML_V2_CUDA_MAX_EVENTS] = { nullptr };

void ggml_v2_init_cublas_legacy() {
    if (g_cublasH == nullptr) {
        // create streams
        for (int i = 0; i < GGML_V2_CUDA_MAX_STREAMS; ++i) {
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams[i], hipStreamNonBlocking));
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams2[i], hipStreamNonBlocking));
        }
        // create events
        for (int i = 0; i < GGML_V2_CUDA_MAX_EVENTS; ++i) {
            CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvents[i], hipEventDisableTiming));
        }

        // create cublas handle
        CUBLAS_CHECK(hipblasCreate(&g_cublasH));
        CUBLAS_CHECK(hipblasSetMathMode(g_cublasH, HIPBLAS_TF32_TENSOR_OP_MATH));

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, nullptr));
    }
}



static hipError_t ggml_v2_cuda_h2d_tensor_2d(void * dst, const struct ggml_v2_tensor * src, uint64_t i3, uint64_t i2, hipStream_t stream) {
    const uint64_t ne0 = src->ne[0];
    const uint64_t ne1 = src->ne[1];
    const uint64_t nb0 = src->nb[0];
    const uint64_t nb1 = src->nb[1];
    const uint64_t nb2 = src->nb[2];
    const uint64_t nb3 = src->nb[3];
    const enum ggml_v2_type type = src->type;
    const size_t ts = ggml_v2_type_size(type);
    const size_t bs = ggml_v2_blck_size(type);

    const void * x = (const void *) ((const char *) src->data + i2*nb2 + i3*nb3);
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst, x, ne1*nb1, hipMemcpyHostToDevice, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst, ts*ne0/bs, x, nb1, ts*ne0/bs, ne1, hipMemcpyHostToDevice, stream);
    } else {
        for (uint64_t i1 = 0; i1 < ne1; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) ((char *) dst + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, hipMemcpyHostToDevice, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

static void ggml_v2_cuda_mul_mat_f32(const ggml_v2_tensor * src0, const ggml_v2_tensor * src1, ggml_v2_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;

    size_t x_size, y_size, d_size;
    float * d_X = (float *) ggml_v2_cuda_pool_malloc(n_mm * sizeof(float) * x_ne, &x_size);
    float * d_Y = (float *) ggml_v2_cuda_pool_malloc(n_mm * sizeof(float) * y_ne, &y_size);
    float * d_D = (float *) ggml_v2_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_V2_CUDA_MAX_STREAMS];

            float * c_X = d_X + i * x_ne;
            float * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;

            // copy data to device
            CUDA_CHECK(ggml_v2_cuda_h2d_tensor_2d(c_X, src0, i03, i02, cudaStream));
            CUDA_CHECK(ggml_v2_cuda_h2d_tensor_2d(c_Y, src1, i03, i02, cudaStream));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, ne00,
                                c_Y, ne10,
                        &beta,  c_D, ne01));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_v2_cuda_pool_free(d_X, x_size);
    ggml_v2_cuda_pool_free(d_Y, y_size);
    ggml_v2_cuda_pool_free(d_D, d_size);
}

static void ggml_v2_cuda_mul_mat_f16(const ggml_v2_tensor * src0, const ggml_v2_tensor * src1, ggml_v2_tensor * dst, void * wdata, size_t /* wsize */) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb10 = src1->nb[0];
    const int nb11 = src1->nb[1];
    const int nb12 = src1->nb[2];
    const int nb13 = src1->nb[3];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;

    size_t x_size, y_size, d_size;
    half  * d_X =  (half *) ggml_v2_cuda_pool_malloc(n_mm * sizeof(half) * x_ne, &x_size);
    half  * d_Y =  (half *) ggml_v2_cuda_pool_malloc(n_mm * sizeof(half) * y_ne, &y_size);
    float * d_D = (float *) ggml_v2_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);

    bool src1_cont_rows = nb10 == sizeof(float);
    bool src1_cont_cols = (size_t)nb11 == ne11*sizeof(float);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_V2_CUDA_MAX_STREAMS];

            half  * c_X = d_X + i * x_ne;
            half  * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;

            // copy src0 to device
            CUDA_CHECK(ggml_v2_cuda_h2d_tensor_2d(c_X, src0, i03, i02, cudaStream));

            // convert src1 to fp16
            // TODO: use multiple threads
            ggml_v2_fp16_t * const tmp = (ggml_v2_fp16_t *) wdata + (ne11 * ne10) * (i03 * ne02 + i02);
            char * src1i = (char *) src1->data + i03*nb13 + i02*nb12;
            if (src1_cont_rows) {
                if (src1_cont_cols) {
                    ggml_v2_fp32_to_fp16_row((float *) src1i, tmp, ne10*ne11);
                }
                else {
                    for (int64_t i01 = 0; i01 < ne11; i01++) {
                        ggml_v2_fp32_to_fp16_row((float *) (src1i + i01*nb11), tmp + i01*ne10, ne10);
                    }
                }
            }
            else {
                for (int64_t i01 = 0; i01 < ne11; i01++) {
                    for (int64_t i00 = 0; i00 < ne10; i00++) {
                        // very slow due to no inlining
                        tmp[i01*ne10 + i00] = ggml_v2_fp32_to_fp16(*(float *) (src1i + i01*nb11 + i00*nb10));
                    }
                }
            }

            // copy src1 to device
            CUDA_CHECK(hipMemcpyAsync(c_Y, tmp, sizeof(half) * y_ne, hipMemcpyHostToDevice, cudaStream));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasGemmEx(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, HIP_R_16F, ne00,
                                c_Y, HIP_R_16F, ne10,
                        &beta,  c_D, HIP_R_32F, ne01,
                        HIPBLAS_COMPUTE_32F_FAST_16F,
                        HIPBLAS_GEMM_DEFAULT));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_v2_cuda_pool_free(d_X, x_size);
    ggml_v2_cuda_pool_free(d_Y, y_size);
    ggml_v2_cuda_pool_free(d_D, d_size);
}

static void ggml_v2_cuda_mul_mat_q_f32(const ggml_v2_tensor * src0, const ggml_v2_tensor * src1, ggml_v2_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];
    const ggml_v2_type type = src0->type;

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;
    const size_t q_sz = ggml_v2_type_size(type) * x_ne / ggml_v2_blck_size(type);

    size_t x_size, y_size, d_size, q_size;
    float * d_X = (float *) ggml_v2_cuda_pool_malloc(n_mm * sizeof(float) * x_ne, &x_size);
    float * d_Y = (float *) ggml_v2_cuda_pool_malloc(n_mm * sizeof(float) * y_ne, &y_size);
    float * d_D = (float *) ggml_v2_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);
    char  * d_Q = (char  *) ggml_v2_cuda_pool_malloc(n_mm * q_sz, &q_size);

    const to_fp32_cuda_t to_fp32_cuda = ggml_v2_get_to_fp32_cuda(type);
    GGML_V2_ASSERT(to_fp32_cuda != nullptr);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_V2_CUDA_MAX_STREAMS];
            hipStream_t cudaStream2 = g_cudaStreams2[i % GGML_V2_CUDA_MAX_STREAMS];
            hipEvent_t  cudaEvent = g_cudaEvents[i % GGML_V2_CUDA_MAX_EVENTS];

            float * c_X = d_X + i * x_ne;
            float * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;
            char  * c_Q = d_Q + i * q_sz;

            // copy src0 and convert to fp32 on device
            CUDA_CHECK(ggml_v2_cuda_h2d_tensor_2d(c_Q, src0, i03, i02, cudaStream2));
            to_fp32_cuda(c_Q, c_X, x_ne, cudaStream2);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventRecord(cudaEvent, cudaStream2));

            // copy src1 to device
            CUDA_CHECK(ggml_v2_cuda_h2d_tensor_2d(c_Y, src1, i03, i02, cudaStream));

            // wait for conversion
            CUDA_CHECK(hipStreamWaitEvent(cudaStream, cudaEvent, 0));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, ne00,
                                c_Y, ne10,
                        &beta,  c_D, ne01));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_v2_cuda_pool_free(d_X, x_size);
    ggml_v2_cuda_pool_free(d_Y, y_size);
    ggml_v2_cuda_pool_free(d_D, d_size);
    ggml_v2_cuda_pool_free(d_Q, q_size);
}

static bool ggml_v2_cuda_mul_mat_use_f16(const struct ggml_v2_tensor * src0, const struct ggml_v2_tensor * src1, struct ggml_v2_tensor * /* dst */) {
    size_t src0_sz = ggml_v2_nbytes(src0);
    size_t src1_sz = ggml_v2_nbytes(src1);

    // mul_mat_q: src0 is converted to fp32 on device
    size_t mul_mat_q_transfer = src0_sz + src1_sz;

    // mul_mat_f16: src1 is converted to fp16 on cpu
    size_t mul_mat_f16_transfer = src0_sz + sizeof(half) * ggml_v2_nelements(src1);

    // choose the smaller one to transfer to the device
    // TODO: this is not always the best choice due to the overhead of converting to fp16
    return mul_mat_f16_transfer < mul_mat_q_transfer;
}

void ggml_v2_cuda_mul_mat_legacy(const ggml_v2_tensor * src0, const ggml_v2_tensor * src1, ggml_v2_tensor * dst, void * wdata, size_t wsize) {
    GGML_V2_ASSERT(ggml_v2_cuda_can_mul_mat(src0, src1, dst));

    if (src0->type == GGML_V2_TYPE_F32) {
        ggml_v2_cuda_mul_mat_f32(src0, src1, dst);
    }
    else if (src0->type == GGML_V2_TYPE_F16) {
        if (ggml_v2_cuda_mul_mat_use_f16(src0, src1, dst)) {
            ggml_v2_cuda_mul_mat_f16(src0, src1, dst, wdata, wsize);
        }
        else {
            ggml_v2_cuda_mul_mat_q_f32(src0, src1, dst);
        }
    }
    else if (ggml_v2_is_quantized(src0->type)) {
        ggml_v2_cuda_mul_mat_q_f32(src0, src1, dst);
    }
    else {
        GGML_V2_ASSERT(false);
    }
}

